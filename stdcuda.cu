#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void bucketMeanKernel(const double* data, double* bucketMeans, int bucketSize, int dataSize) {
    int bucketIndex = blockIdx.x;
    int startIdx = bucketIndex * bucketSize;
    int endIdx = min(startIdx + bucketSize, dataSize);

    double sum = 0.0;
    for (int i = startIdx; i < endIdx; ++i) {
        sum += data[i];
    }
    bucketMeans[bucketIndex] = sum / (endIdx - startIdx);
}

__global__ void bucketVarianceKernel(const double* data, const double* bucketMeans, double* bucketVariances, int bucketSize, int dataSize) {
    int bucketIndex = blockIdx.x;
    int startIdx = bucketIndex * bucketSize;
    int endIdx = min(startIdx + bucketSize, dataSize);

    double sum = 0.0;
    for (int i = startIdx; i < endIdx; ++i) {
        double diff = data[i] - bucketMeans[bucketIndex];
        sum += diff * diff;
    }
    bucketVariances[bucketIndex] = sum / (endIdx - startIdx);
}

// Prepare the dataset by randomly generating data points
double* prepare(int dataSize) {
    double* data = new double[dataSize];
    for (int i = 0; i < dataSize; ++i) {
        data[i] = static_cast<double>(rand()) / RAND_MAX * 100.0; // Random numbers between 0 and 100
    }
    double* d_data;
    hipMalloc(&d_data, dataSize * sizeof(double));
    hipMemcpy(d_data, data, dataSize * sizeof(double), hipMemcpyHostToDevice);
    delete[] data; // Free the host memory
    return d_data;
}

// Compute the standard deviation
double compute(const double* data, int dataSize, int threadsPerBlock, float &elapsedTime, int &bucketSize, int &numberOfBuckets) {
    bucketSize = 16; // We adjust appropriate bucket size
    numberOfBuckets = (dataSize + bucketSize - 1) / bucketSize;
    
    double* d_bucketMeans, *d_bucketVariances;
    double* h_bucketMeans = new double[numberOfBuckets];
    double* h_bucketVariances = new double[numberOfBuckets];

    hipMalloc(&d_bucketMeans, numberOfBuckets * sizeof(double));
    hipMalloc(&d_bucketVariances, numberOfBuckets * sizeof(double));

    // Start timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernels
    bucketMeanKernel<<<numberOfBuckets, threadsPerBlock>>>(data, d_bucketMeans, bucketSize, dataSize);
    hipDeviceSynchronize();

    bucketVarianceKernel<<<numberOfBuckets, threadsPerBlock>>>(data, d_bucketMeans, d_bucketVariances, bucketSize, dataSize);
    hipDeviceSynchronize();

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy results back to host
    hipMemcpy(h_bucketMeans, d_bucketMeans, numberOfBuckets * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_bucketVariances, d_bucketVariances, numberOfBuckets * sizeof(double), hipMemcpyDeviceToHost);

    // Final calculation of variance on host
    double variance = 0.0;
    for (int i = 0; i < numberOfBuckets; ++i) {
        variance += h_bucketVariances[i];
    }
    variance /= numberOfBuckets;

    // Clean up
    hipFree(d_bucketMeans);
    hipFree(d_bucketVariances);
    delete[] h_bucketMeans;
    delete[] h_bucketVariances;

    // Return the standard deviation
    return sqrt(variance);
}

// Report the results including bucket size and number of buckets
void report(int threadsPerBlock, double stdDev, float elapsedTime, int bucketSize, int numberOfBuckets) {
    cout << "Calculating standard deviation with " << threadsPerBlock << " threads per block." << endl;
    cout << "Bucket Size: " << bucketSize << ", Number of Buckets: " << numberOfBuckets << endl;
    cout << "Standard Deviation (CUDA): " << stdDev << endl;
    cout << "Execution Time (Threads Per Block: " << threadsPerBlock << "): " << elapsedTime << " ms" << endl << endl;
}

int main(int argc, char *argv[]) {
    // Check if the dataSize argument is provided
    if (argc != 2) {
        cerr << "Usage: " << argv[0] << " <dataSize>" << endl;
        return 1; // Return an error code
    }

    // Convert the argument to an integer
    int dataSize = atoi(argv[1]);
    if (dataSize <= 0) {
        cerr << "Error: dataSize must be a positive integer." << endl;
        return 1; // Return an error code
    }

    double* data = prepare(dataSize);

    vector<int> threadsPerBlockConfigs = {32, 64, 96, 128};
    for (int threadsPerBlock : threadsPerBlockConfigs) {
        float elapsedTime;
        int bucketSize, numberOfBuckets;
        double stdDev = compute(data, dataSize, threadsPerBlock, elapsedTime, bucketSize, numberOfBuckets);
        report(threadsPerBlock, stdDev, elapsedTime, bucketSize, numberOfBuckets);
    }

    hipFree(data); // Free the allocated memory
    return 0;
}