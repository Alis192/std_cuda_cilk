#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

using namespace std;

__global__ void bucketMeanKernel(const double* data, double* bucketMeans, int bucketSize, int dataSize) {
    int bucketIndex = blockIdx.x;
    int startIdx = bucketIndex * bucketSize;
    int endIdx = min(startIdx + bucketSize, dataSize);

    double sum = 0.0;
    for (int i = startIdx; i < endIdx; ++i) {
        sum += data[i];
    }
    bucketMeans[bucketIndex] = sum / (endIdx - startIdx);
}

__global__ void bucketVarianceKernel(const double* data, const double* bucketMeans, double* bucketVariances, int bucketSize, int dataSize) {
    int bucketIndex = blockIdx.x;
    int startIdx = bucketIndex * bucketSize;
    int endIdx = min(startIdx + bucketSize, dataSize);

    double sum = 0.0;
    for (int i = startIdx; i < endIdx; ++i) {
        double diff = data[i] - bucketMeans[bucketIndex];
        sum += diff * diff;
    }
    bucketVariances[bucketIndex] = sum / (endIdx - startIdx);
}

// Prepare the dataset by randomly generating data points
double* prepareUnified(int dataSize) {
    double* data;
    hipMallocManaged(&data, dataSize * sizeof(double));
    for (int i = 0; i < dataSize; ++i) {
        data[i] = static_cast<double>(rand()) / RAND_MAX * 100.0; // Random numbers between 0 and 100
    }
    return data;
}

// Compute the standard deviation
double compute(const double* data, int dataSize, int threadsPerBlock, float &elapsedTime, int &bucketSize, int &numberOfBuckets) {
    bucketSize = 16; // We adjust appropriate bucket size
    numberOfBuckets = (dataSize + bucketSize - 1) / bucketSize;
    
    double* d_bucketMeans, *d_bucketVariances;
    double* h_bucketMeans = new double[numberOfBuckets];
    double* h_bucketVariances = new double[numberOfBuckets];

    hipMalloc(&d_bucketMeans, numberOfBuckets * sizeof(double));
    hipMalloc(&d_bucketVariances, numberOfBuckets * sizeof(double));

    // Start timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernels
    bucketMeanKernel<<<numberOfBuckets, threadsPerBlock>>>(data, d_bucketMeans, bucketSize, dataSize);
    hipDeviceSynchronize();

    bucketVarianceKernel<<<numberOfBuckets, threadsPerBlock>>>(data, d_bucketMeans, d_bucketVariances, bucketSize, dataSize);
    hipDeviceSynchronize();

    // Stop timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy results back to host
    hipMemcpy(h_bucketMeans, d_bucketMeans, numberOfBuckets * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_bucketVariances, d_bucketVariances, numberOfBuckets * sizeof(double), hipMemcpyDeviceToHost);

    // Final calculation of variance on host
    double variance = 0.0;
    for (int i = 0; i < numberOfBuckets; ++i) {
        variance += h_bucketVariances[i];
    }
    variance /= numberOfBuckets;

    // Clean up
    hipFree(d_bucketMeans);
    hipFree(d_bucketVariances);
    delete[] h_bucketMeans;
    delete[] h_bucketVariances;

    // Return the standard deviation
    return sqrt(variance);
}

// Report the results including bucket size and number of buckets
void report(int threadsPerBlock, double stdDev, float elapsedTime, int bucketSize, int numberOfBuckets) {
    cout << "Calculating standard deviation with " << threadsPerBlock << " threads per block." << endl;
    cout << "Bucket Size: " << bucketSize << ", Number of Buckets: " << numberOfBuckets << endl;
    cout << "Standard Deviation (CUDA): " << stdDev << endl;
    cout << "Execution Time (Threads Per Block: " << threadsPerBlock << "): " << elapsedTime << " ms" << endl << endl;
}


void calculateMeanAndStdDevSequentially(const double* data, int dataSize, double &mean, double &stdDev) {
    double sum = 0.0;
    for (int i = 0; i < dataSize; ++i) {
        sum += data[i];
    }
    mean = sum / dataSize;

    double varianceSum = 0.0;
    for (int i = 0; i < dataSize; ++i) {
        varianceSum += (data[i] - mean) * (data[i] - mean);
    }
    double variance = varianceSum / dataSize;
    stdDev = sqrt(variance);
}


int main(int argc, char *argv[]) {
    // Check if the dataSize argument is provided
    if (argc != 2) {
        cerr << "Usage: " << argv[0] << " <dataSize>" << endl;
        return 1; // Return an error code
    }

    // Convert the argument to an integer
    int dataSize = atoi(argv[1]);
    if (dataSize <= 0) {
        cerr << "Error: dataSize must be a positive integer." << endl;
        return 1; // Return an error code
    }

    // Prepare the dataset with Unified Memory
    double* data = prepareUnified(dataSize);

    // Calculate mean and standard deviation sequentially for correctness checking
    double correctMean, correctStdDev;
    calculateMeanAndStdDevSequentially(data, dataSize, correctMean, correctStdDev);
    cout << "Sequential Standard Deviation: " << correctStdDev << endl;

    // Proceed with CUDA computations
    vector<int> threadsPerBlockConfigs = {32, 64, 96, 128};
    for (int threadsPerBlock : threadsPerBlockConfigs) {
        float elapsedTime;
        int bucketSize, numberOfBuckets;
        double stdDev = compute(data, dataSize, threadsPerBlock, elapsedTime, bucketSize, numberOfBuckets);
        report(threadsPerBlock, stdDev, elapsedTime, bucketSize, numberOfBuckets);
        
        // Compare CUDA-computed stdDev with correctStdDev
        cout << "Difference between Sequential and CUDA-computed StdDev: " << abs(correctStdDev - stdDev) << endl;
    }

    // Clean up
    hipFree(data); // Free the allocated unified memory

    return 0;
}
